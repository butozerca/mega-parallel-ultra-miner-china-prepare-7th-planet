#include "hip/hip_runtime.h"
#include<cstdio>
#include"sha256.hpp"

extern "C" {
__global__ 
void Gpu_hash(const char* input, int length, int nonce_offset, int difficulty, int* result)
{
    int nonce=blockIdx.x*blockDim.x+threadIdx.x;
    char nonce_input[80];
    memcpy(nonce_input, input, length);
    memcpy(nonce_input+length, (void*)&nonce, 4);
    
    unsigned char digest[SHA256::DIGEST_SIZE];
 
    SHA256 ctx = SHA256();
    ctx.init();
    ctx.update(nonce_input, length + 4);
    ctx.final(digest);

    ctx.init();
    ctx.update(digest, 32);
    ctx.final(nonce_input);
    
    for (int i = 0; i < (difficulty >> 3); ++i)
        if (nonce_input[i] != 0) return;
    if (nonce_input[difficulty >> 3] <= 255 >> difficulty & 7)
        *result = nonce;
}
}
