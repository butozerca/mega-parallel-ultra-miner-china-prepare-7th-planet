#include "hip/hip_runtime.h"
#include<cstdio>
#include"sha256.hpp"

extern "C" {
__global__ 
void Gpu_hash(const char* input, int length, int nonce_offset, int difficulty, int* result)
{
    int nonce=blockIdx.x*blockDim.x+threadIdx.x;
    char nonce_input[length+4];
    memcpy(nonce_input, input, length);
    memcpy(nonce_input+length, (void*)&nonce, 4);
    
    unsigned char digest[SHA256::DIGEST_SIZE];
    memset(digest,0,SHA256::DIGEST_SIZE);
 
    SHA256 ctx = SHA256();
    ctx.init();
    ctx.update(input, length);
    ctx.final(digest);
    
    
    unsigned char digest2[SHA256::DIGEST_SIZE];
    memset(digest2,0,SHA256::DIGEST_SIZE);
 
    SHA256 ctx = SHA256();
    ctx.init();
    ctx.update(input, length);
    ctx.final(digest2);
    
    for (int i = 0; i < (difficulty >> 1); ++i){
        
    }
}
}